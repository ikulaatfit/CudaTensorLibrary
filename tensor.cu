#ifndef TENSOR_CU_H
#define TENSOR_CU_H
#include <mma.h>

#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif

namespace CudaTensorLib
{
    template <typename T> class componentTypes;

    template <> class componentTypes<unsigned char>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 1;};
        typedef unsigned char t;
        typedef unsigned char t1;
        typedef uchar2 t2;
    };

    template <> class componentTypes<uchar2>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 2;};
        typedef uchar2 t;
        typedef unsigned char t1;
        typedef uchar2 t2;
    };

    template <> class componentTypes<uchar4>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 4;};
        typedef uchar4 t;
        typedef unsigned char t1;
        typedef uchar2 t2;
    };

    template <> class componentTypes<half>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 1;};
        typedef half t;
        typedef half t1;
        typedef half2 t2;
    };

    template <> class componentTypes<half2>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 2;};
        typedef half2 t;
        typedef half t1;
        typedef half2 t2;
    };

    template <> class componentTypes<int>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 1;};
        typedef int t;
        typedef int t1;
        typedef int2 t2;
    };

    template <> class componentTypes<int2>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 2;};
        typedef int2 t;
        typedef int t1;
        typedef int2 t2;
    };

    template <> class componentTypes<int4>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 4;};
        typedef int4 t;
        typedef int t1;
        typedef int2 t2;
    };

    template <> class componentTypes<float>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 1;};
        typedef float t;
        typedef float t1;
        typedef float2 t2;
    };

    template <> class componentTypes<float2>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 2;};
        typedef float2 t;
        typedef float t1;
        typedef float2 t2;
    };

    template <> class componentTypes<float4>
    {
    public:
        static constexpr __device__ int getNumComponents(){return 4;};
        typedef float4 t;
        typedef float t1;
        typedef float2 t2;
    };

    template <typename mat_type, int M, int N, int K, typename T> class fragment
    {
    public:
        typedef componentTypes<T> t;
        static constexpr __device__ int getNumElementsPerComponentWithoutCopy() { return (std::is_same<mat_type, nvcuda::wmma::matrix_a>::value ? 1 : N) * (std::is_same<mat_type, nvcuda::wmma::matrix_b>::value ? 1 : M) * (std::is_same<mat_type, nvcuda::wmma::accumulator>::value ? 1 : K) / 32; };
        static constexpr __device__ int getNumElementsPerComponent() { return (std::is_same<typename t::t1, half>::value && (M * N == 256) && (!std::is_same<mat_type, nvcuda::wmma::accumulator>::value)) ? 16 : getNumElementsPerComponentWithoutCopy(); };
        static constexpr __device__ int getCopyPerThread() { return getNumElementsPerComponent() / getNumElementsPerComponentWithoutCopy(); };
        static constexpr __device__ int getNumComponents(){return componentTypes<T>::getNumComponents();};
        __device__ inline void fill(T data);
        typename t::t1 x[t::getNumComponents()][getNumElementsPerComponent()];
    };

    template<typename mat_type, int M, int N, int K, typename T>
    __device__ inline void fragment<mat_type, M, N, K, T>::fill(T data)
    {
#pragma unroll
        for (int element_id = 0; element_id < getNumComponents(); element_id++)
        {
#pragma unroll
            for (int component_id = 0; component_id < getNumElementsPerComponent(); component_id++)
            {
                x[element_id][component_id] = data;
            }
        }
    };

template <typename T, class DerivedClass>
class BaseManipulator
{
    public:
        __device__ inline BaseManipulator(){}
        using t = componentTypes<T>;

        template <typename regT, bool horizontalDirection, int regStride>
        __device__ inline void loadNxData2D(int x, int y, regT *reg)
        {
            typename t::t tmp_data = (horizontalDirection) ? ((DerivedClass *)this)->template loadData2D<typename t::t>(x, y) : ((DerivedClass *)this)->template loadData2D<typename t::t>(y, x);
            #pragma unroll
            for(unsigned int element_id = 0; element_id < t::getNumComponents(); element_id++)
            {
                ((typename componentTypes<regT>::t1 *)reg)[regStride * element_id] = (typename componentTypes<regT>::t1)(((typename t::t1 *)&tmp_data)[element_id]);
            }
        }
    
        template <typename regT, bool horizontalDirection, int regStride>
        __device__ inline void storeNxData2D(int x, int y, regT *reg)
        {
            typename t::t tmp_data;
            #pragma unroll
            for(unsigned int element_id = 0; element_id < t::getNumComponents(); element_id++)
            {
                ((typename t::t1 *)&tmp_data)[element_id] = (typename t::t1)(((typename componentTypes<regT>::t1 *)reg)[regStride * element_id]);
            }
    
            (horizontalDirection) ? ((DerivedClass *)this)->template storeData2D<typename t::t>(tmp_data, x, y) : ((DerivedClass *)this)->template storeData2D<typename t::t>(tmp_data, y, x);
        }
    
        template <typename regT, bool horizontalDirection>
        __device__ inline void loadNx4BData2D(int x, int y, regT *reg)
        {
            constexpr unsigned int vals_per_reg = 4/sizeof(typename componentTypes<regT>::t1);
            #pragma unroll
            for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
            {
                this->template loadNxData2D<regT, horizontalDirection, vals_per_reg>(x + stride_x, y, reg + stride_x);
            }
        }
    
        template <typename regT, bool horizontalDirection>
        __device__ inline void storeNx4BData2D(int x, int y, regT *reg)
        {
            constexpr unsigned int vals_per_reg = 4/sizeof(typename componentTypes<regT>::t1);
            #pragma unroll
            for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
            {
                this->template storeNxData2D<regT, horizontalDirection, vals_per_reg>(x + stride_x, y, reg + stride_x);
            }
        }
    
        template <typename regT, bool horizontalDirection, int elementsCount>
        __device__ inline void loadNxMData2D(int x, int y, regT *reg)
        {
            constexpr unsigned int vals_per_reg = elementsCount;
            #pragma unroll
            for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
            {
                this->template loadNxData2D<regT, horizontalDirection, vals_per_reg>(x + stride_x, y, reg + stride_x);
            }
        }
    
        template <typename regT, bool horizontalDirection, int elementsCount>
        __device__ inline void storeNxMData2D(int x, int y, regT *reg)
        {
            constexpr unsigned int vals_per_reg = elementsCount;
            #pragma unroll
            for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
            {
                this->template storeNxData2D<regT, horizontalDirection, vals_per_reg>(x + stride_x, y, reg + stride_x);
            }
        }
};

template <typename T, class ParentDerivedClassT = void>
class Linear2DManipulator : public BaseManipulator<T, typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Linear2DManipulator<T>, ParentDerivedClassT>::type>
{
public:
    using Linear2DDerivedClassT = typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Linear2DManipulator<T>, ParentDerivedClassT>::type;
    __device__ inline Linear2DManipulator(T *ptr, int x, int y, int stride): BaseManipulator<T,Linear2DDerivedClassT>(), ptr(ptr + x + y * stride), stride(stride){}
    int stride;
    T *ptr;

    template <typename outT>
    __device__ inline outT loadData2D(int x, int y = 0)
    {
        return (outT)(*((Linear2DDerivedClassT *)this)->getPtr2D(x, y));
    }

    template <typename inT>
    __device__ inline void storeData2D(inT data, int x, int y = 0)
    {
        (*((Linear2DDerivedClassT *)this)->getPtr2D(x, y)) = (typename BaseManipulator<T, Linear2DDerivedClassT>::t::t)data;
    }

    __device__ inline T *getPtr2D(int x, int y = 0)
    {
        return this->ptr + x + y * this->stride;
    }

    __device__ inline unsigned int load4BData2D(int x, int y = 0)
    {
        return ((unsigned int *)(((Linear2DDerivedClassT *)this)->getPtr2D(x,y)))[0];
    }


    __device__ inline void store4BData2D(unsigned int data, int x, int y = 0)
    {
        ((unsigned int *)(((Linear2DDerivedClassT *)this)->getPtr2D(x,y)))[0] = data;
    }

    __device__ inline typename BaseManipulator<T, Linear2DDerivedClassT>::t::t2 load2Data2D(int x, int y = 0)
    {
        return ((typename BaseManipulator<T, Linear2DDerivedClassT>::t::t2 *)(((Linear2DDerivedClassT *)this)->getPtr2D(x,y)))[0];
    }

    __device__ inline void store2Data2D(typename BaseManipulator<T, Linear2DDerivedClassT>::t::t2 data, int x, int y = 0)
    {
        ((typename BaseManipulator<T, Linear2DDerivedClassT>::t::t2 *)(((Linear2DDerivedClassT *)this)->getPtr2D(x,y)))[0] = data;
    }
};

enum class DimensionMapping3To2 { XY, XZ, YZ };

template <typename T, DimensionMapping3To2 dim_mapping, class ParentDerivedClassT = void>
class Linear3DManipulator : public Linear2DManipulator<T, typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Linear3DManipulator<T, dim_mapping>, ParentDerivedClassT>::type>
{
public:
    using Linear3DDerivedClassT = typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Linear3DManipulator<T, dim_mapping>, ParentDerivedClassT>::type;
    __device__ inline Linear3DManipulator(T *ptr, int x, int y, int z, int stride, int pitch): Linear2DManipulator<T, Linear3DDerivedClassT>(ptr + z * pitch, x, y, stride), pitch(pitch){}
    int pitch;

    template <typename outT>
    __device__ inline outT loadData3D(int x, int y = 0, int z = 0)
    {
        return (outT)(*this->getPtr3D(x, y, z));
    }

    template <typename inT>
    __device__ inline void storeData3D(inT data, int x = 0, int y = 0, int z = 0)
    {
        (*this->getPtr(x, y, z)) = (typename BaseManipulator<T, Linear3DDerivedClassT>::t::t)data;
    }

    __device__ inline unsigned int load4BData3D(int x = 0, int y = 0, int z = 0)
    {
        return ((unsigned int *)(this->getPtr3D(x,y,z)))[0];
    }

    __device__ inline void store4BData3D(unsigned int data, int x = 0, int y = 0, int z = 0)
    {
        ((unsigned int *)(this->getPtr3D(x,y,z)))[0] = data;
    }

    __device__ inline typename BaseManipulator<T, Linear3DDerivedClassT>::t::t2 load2Data3D(int x = 0, int y = 0, int z = 0)
    {
        return ((typename BaseManipulator<T, Linear3DDerivedClassT>::t::t2 *)(this->getPtr3D(x,y,z)))[0];
    }

    __device__ inline void store2Data3D(typename BaseManipulator<T, Linear3DDerivedClassT>::t::t2 data, int x = 0, int y = 0, int z = 0)
    {
        ((typename BaseManipulator<T, Linear3DDerivedClassT>::t::t2 *)(this->getPtr3D(x,y,z)))[0] = data;
    }

    __device__ inline int getXFrom2D(int id0, int id1)
    {
        return ((dim_mapping == DimensionMapping3To2::XY) || (dim_mapping == DimensionMapping3To2::XZ)) ? id0 : 0;
    }

    __device__ inline int getYFrom2D(int id0, int id1)
    {
        return (dim_mapping == DimensionMapping3To2::XY) ? id1 : ((dim_mapping == DimensionMapping3To2::YZ) ? id0 : 0);
    }

    __device__ inline int getZFrom2D(int id0, int id1)
    {
        return ((dim_mapping == DimensionMapping3To2::XZ) || (dim_mapping == DimensionMapping3To2::YZ)) ? id1 : 0;
    }

    __device__ inline T *getPtr2D(int id0 = 0, int id1 = 0)
    {
        return Linear2DManipulator<T, Linear3DDerivedClassT>(getXFrom2D(id0, id1), getYFrom2D(id0, id1)) + getZFrom2D(id0, id1) * this->pitch;
    }
};


template <typename T, typename ParentDerivedClassT = void>
class Block2DManipulator : public BaseManipulator<T, typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Block2DManipulator<T>, ParentDerivedClassT>::type>
{
public:
    using Block2DDerivedClassT = typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Block2DManipulator<T>, ParentDerivedClassT>::type;
    __device__ inline Block2DManipulator(int x, int y) : BaseManipulator<T, Block2DDerivedClassT>(), x(x), y(y){}
    int x;
    int y;
};

template <typename T, DimensionMapping3To2 dim_mapping, typename ParentDerivedClassT = void>
class Block3DManipulator : public Block2DManipulator<T, typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Block3DManipulator<T, dim_mapping>, ParentDerivedClassT>::type>
{
public:
    using Block3DDerivedClassT = typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Block3DManipulator<T, dim_mapping>, ParentDerivedClassT>::type;
    __device__ inline Block3DManipulator(int x, int y, int z) : Block2DManipulator<T, Block3DDerivedClassT>(x, y), z(z){}
    int z;

    __device__ inline int getXFrom2D(int id0, int id1)
    {
        return ((dim_mapping == DimensionMapping3To2::XY) || (dim_mapping == DimensionMapping3To2::XZ)) ? id0 : 0;
    }

    __device__ inline int getYFrom2D(int id0, int id1)
    {
        return (dim_mapping == DimensionMapping3To2::XY) ? id1 : ((dim_mapping == DimensionMapping3To2::YZ) ? id0 : 0);
    }

    __device__ inline int getZFrom2D(int id0, int id1)
    {
        return ((dim_mapping == DimensionMapping3To2::XZ) || (dim_mapping == DimensionMapping3To2::YZ)) ? id1 : 0;
    }

    template <typename outT>
    __device__ inline outT loadData2D(int id0 = 0, int id1 = 0)
    {
        return ((Block3DDerivedClassT *)this)->template loadData3D<outT>(this->getXFrom2D(id0, id1), this->getYFrom2D(id0, id1), this->getZFrom2D(id0, id1));
    }

    template <typename inT>
    __device__ inline void storeData2D(inT data, int id0 = 0, int id1 = 0)
    {
        return ((Block3DDerivedClassT *)this)->template storeData2D<inT>(data, this->getXFrom2D(id0, id1), this->getYFrom2D(id0, id1), this->getZFrom2D(id0, id1));
    }

    __device__ inline unsigned int load4BData2D(int id0 = 0, int id1 = 0)
    {
        return ((Block3DDerivedClassT *)this)->load4BData3D(this->getXFrom2D(id0, id1), this->getYFrom2D(id0, id1), this->getZFrom2D(id0, id1));
    }

    __device__ inline void store4BData2D(unsigned int data, int id0 = 0, int id1 = 0)
    {
        ((Block3DDerivedClassT *)this)->store4BData3D(data, this->getXFrom2D(id0, id1), this->getYFrom2D(id0, id1), this->getZFrom2D(id0, id1));
    }

    __device__ inline typename BaseManipulator<T, Block3DDerivedClassT>::t::t2 load2Data2D(int id0 = 0, int id1 = 0)
    {
        return ((Block3DDerivedClassT *)this)->load2Data3D(this->getXFrom2D(id0, id1), this->getYFrom2D(id0, id1), this->getZFrom2D(id0, id1));
    }

    __device__ inline void store2Data2D(typename BaseManipulator<T, Block3DDerivedClassT>::t::t2 data, int id0 = 0, int id1 = 0)
    {
        ((Block3DDerivedClassT *)this)->store2Data3D(data, this->getXFrom2D(id0, id1), this->getYFrom2D(id0, id1), this->getZFrom2D(id0, id1));
    }
};


template <typename T, DimensionMapping3To2 dim_mapping, typename ParentDerivedClassT = void>
class Surface3DWithOffsetsManipulator : public Block3DManipulator<T, dim_mapping, typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Surface3DWithOffsetsManipulator<T, dim_mapping>, ParentDerivedClassT>::type>
{
public:
    using Surface3DWithOffsetsDerivedClassT = typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Surface3DWithOffsetsManipulator<T, dim_mapping>, ParentDerivedClassT>::type;

    __device__ Surface3DWithOffsetsManipulator(hipSurfaceObject_t *buffer, int2 *offsets, int x, int y, int z): Block3DManipulator<T, dim_mapping, Surface3DWithOffsetsDerivedClassT>(x, y, z), buffer(buffer), offsets(offsets){}

    __device__ inline unsigned int load4BData3D(int x, int y = 0, int z = 0)
    {
        constexpr unsigned int vals_per_reg = 4/sizeof(typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1);
        unsigned int data;
        #pragma unroll
        for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
        {
            ((typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1 *)&data)[stride_x] = ((Surface3DWithOffsetsDerivedClassT *)this)->template loadData3D<typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1>(x + stride_x, y, z);
        }
        return data;
    }

    __device__ inline void store4BData3D(unsigned int data, int x, int y = 0, int z = 0)
    {
        constexpr unsigned int vals_per_reg = 4/sizeof(typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1);
        #pragma unroll
        for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
        {
            ((Surface3DWithOffsetsDerivedClassT *)this)->template storeData3D<typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1>(((typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1 *)&data)[stride_x], x + stride_x, y, z);
        }
    }

    __device__ inline typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t2 load2Data3D(int x, int y = 0, int z = 0)
    {
        constexpr unsigned int vals_per_reg = 2;
        typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t2 data;
        #pragma unroll
        for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
        {
            ((typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1 *)&data)[stride_x] = ((Surface3DWithOffsetsDerivedClassT *)this)->template loadData3D<typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1>(x + stride_x, y, z);
        }
        return data;
    }

    __device__ inline void store2Data3D(typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t2 data, int x, int y = 0, int z = 0)
    {
        constexpr unsigned int vals_per_reg = 2;
        #pragma unroll
        for(unsigned int stride_x = 0; stride_x < vals_per_reg; stride_x++)
        {
            ((Surface3DWithOffsetsDerivedClassT *)this)->template storeData3D<typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1>(((typename BaseManipulator<T, Surface3DWithOffsetsDerivedClassT>::t::t1 *)&data)[stride_x], x + stride_x, y, z);
        }
    }

    hipSurfaceObject_t *buffer;
    int2 *offsets;
};


template <typename T, DimensionMapping3To2 dim_mapping, typename ParentDerivedClassT = void>
class Layered3DSurfaceWithOffsetsManipulator : public Surface3DWithOffsetsManipulator<T, dim_mapping, typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Layered3DSurfaceWithOffsetsManipulator<T, dim_mapping>, ParentDerivedClassT>::type>
{
public:
    using Layered3DSurfaceWithOffsetsDerivedClassT = typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Layered3DSurfaceWithOffsetsManipulator<T, dim_mapping>, ParentDerivedClassT>::type;

    __device__ Layered3DSurfaceWithOffsetsManipulator(hipSurfaceObject_t *buffer, int2 *offsets, int x, int y, int z): Surface3DWithOffsetsManipulator<T, dim_mapping, Layered3DSurfaceWithOffsetsDerivedClassT>(buffer, offsets, x, y, z){}
    template <typename outT>
    __device__ inline outT loadData3D(int x = 0, int y = 0, int z = 0)
    {
        return surf2DLayeredread<outT>(this->buffer, this->offsets[z].x + this->x + x, this->offsets[z].y + this->y + y, this->z + z, hipBoundaryModeClamp);
    }

    template <typename inT>
    __device__ inline void storeData3D(inT data, int x = 0, int y = 0, int z = 0)
    {
        surf2DLayeredwrite<inT>(data, this->buffer, this->offsets[z].x + this->x + x, this->offsets[z].y + this->y + y, this->z + z);
    }
};

template <typename T, DimensionMapping3To2 dim_mapping, typename ParentDerivedClassT = void>
class Array3DSurfaceWithOffsetsManipulator : public Surface3DWithOffsetsManipulator<T, std::conditional<std::is_same<ParentDerivedClassT, void>::value, Array3DSurfaceWithOffsetsManipulator<T, dim_mapping>, ParentDerivedClassT>::type>
{
public:
    using Array3DSurfaceWithOffsetsDerivedClassT = typename std::conditional<std::is_same<ParentDerivedClassT, void>::value, Array3DSurfaceWithOffsetsManipulator<T, dim_mapping>, ParentDerivedClassT>::type;
    
    __device__ Array3DSurfaceWithOffsetsManipulator(hipSurfaceObject_t *buffer, int2 *offsets, int x, int y, int z): Surface3DWithOffsetsManipulator<T, dim_mapping, Array3DSurfaceWithOffsetsDerivedClassT>(buffer, offsets, x, y, z){}
    template <typename outT>
    __device__ inline outT loadData3D(int x = 0, int y = 0, int z = 0)
    {
        return surf2Dread<outT>(this->buffer + this->z + z, this->offsets[z].x + this->x + x, this->offsets[z].y + this->y + y, hipBoundaryModeClamp);
    }

    template <typename inT>
    __device__ inline void storeData3D(inT data, int x = 0, int y = 0, int z = 0)
    {
        surf2Dwrite<inT>(data, this->buffer + this->z + z, this->offsets[z].x + this->x + x, this->offsets[z].y + this->y + y);
    }
};


// load matrix a
template <typename mat_type, typename inT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_a>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
load_matrix_sync(matT *mat, inT *data, unsigned int stride)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 4/sizeof(matT);
    int val_stride = stride/MAT_B_X_VALUES_PER_REGISTER;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = K/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = M/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;
    constexpr unsigned int copy_stride = MAT_B_Y_LOADS_PER_THREAD*MAT_B_X_LOADS_PER_THREAD;
    constexpr unsigned int copy_per_thread = (std::is_same<half,matT>::value && (K == 16) && (M * N==256)) ? (8/copy_stride) : 1;

    // load from matrix b to fragments - indexing hell
    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = (y_stride_id & 1) + x_stride_id * 2 + (y_stride_id >> 1) * MAT_B_X_LOADS_PER_THREAD * 2;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;
            unsigned int act_data;
            // fast path -> no conversion of datatypes or layouts for threads is needed
            if(std::is_same<inT,matT>::value && std::is_same<MAJOR,nvcuda::wmma::row_major>::value)
            {
                act_data = ((unsigned int *)data)[act_x + act_y * val_stride];
            }
            else
            {
                #pragma unroll
                for(unsigned int x_offset = 0; x_offset < MAT_B_X_VALUES_PER_REGISTER; x_offset++)
                {
                    unsigned int load_id = (std::is_same<MAJOR,nvcuda::wmma::row_major>::value) ? (x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER + act_y * stride) : ((x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER) * stride + act_y);
                    ((matT *)(&act_data))[x_offset] = (matT)(data[load_id]);
                }
            }

            #pragma unroll
            for(unsigned int copy_id = 0; copy_id < copy_per_thread; copy_id++)
            {
                ((unsigned int *)mat)[id + copy_id * copy_stride] = act_data;
            }
        }   
    }
}

template <typename mat_type, class DataManipulatorT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_a>::value, void>::type
load_matrix_sync(fragment<mat_type, M, N, K, matT> &mat, DataManipulatorT &data)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 4/sizeof(typename componentTypes<matT>::t1);
    //int val_stride = stride/MAT_B_X_VALUES_PER_REGISTER;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = K/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = M/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;
    constexpr unsigned int copy_stride = MAT_B_Y_LOADS_PER_THREAD*MAT_B_X_LOADS_PER_THREAD;
    constexpr unsigned int copy_per_thread = (std::is_same<half,matT>::value && (K == 16) && (M * N==256)) ? (8/copy_stride) : 1;
    constexpr bool is_load_linear = std::is_same<typename DataManipulatorT::t::t1,matT>::value && std::is_same<MAJOR,nvcuda::wmma::row_major>::value;

    // load from matrix b to fragments - indexing hell
    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = (y_stride_id & 1) + x_stride_id * 2 + (y_stride_id >> 1) * MAT_B_X_LOADS_PER_THREAD * 2;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;
            unsigned int act_data[mat.getNumComponents()];
            // fast path -> no conversion of datatypes or layouts for threads is needed
            if(is_load_linear)
            //if(false)
            {
                act_data[0] = data.load4BData2D(act_x * MAT_B_X_VALUES_PER_REGISTER, act_y);
            }
            else
            {
                data.template loadNx4BData2D<matT, std::is_same<MAJOR, nvcuda::wmma::row_major>::value>(act_x * MAT_B_X_VALUES_PER_REGISTER, act_y, (typename componentTypes<matT>::t1 *)act_data);
            }

            #pragma unroll
            for(unsigned int element_id = 0; element_id < mat.getNumComponents(); element_id++)
            {
                #pragma unroll
                for(unsigned int copy_id = 0; copy_id < copy_per_thread; copy_id++)
                {
                    ((unsigned int *)(mat.x[element_id]))[id + copy_id * copy_stride] = act_data[element_id];
                }
            }
        }   
    }
}

template <typename mat_type, typename inT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_a>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
load_matrix_sync(CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat, inT *data, unsigned int stride)
{
    load_matrix_sync<mat_type, inT, matT, M, N, K, MAJOR>(mat.x[0], data, stride);
}

/*template <typename mat_type, class DataManipulatorT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_a>::value, void>::type
load_matrix_sync(CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat, DataManipulatorT &data_getter)
{
    load_matrix_sync<mat_type, DataManipulatorT, matT, M, N, K, MAJOR>(mat, data_getter);
}*/

// load matrix b
template <typename mat_type, typename inT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_b>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
load_matrix_sync(matT *mat, inT *data, unsigned int stride)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 4/sizeof(matT);
    int val_stride = stride/MAT_B_X_VALUES_PER_REGISTER;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = K/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = N/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;
    constexpr unsigned int copy_stride = MAT_B_Y_LOADS_PER_THREAD*MAT_B_X_LOADS_PER_THREAD;
    constexpr unsigned int copy_per_thread = (std::is_same<half,matT>::value && (K == 16) && (M * N==256)) ? (8/copy_stride) : 1;

    // load from matrix a to fragments - indexing hell
    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = x_stride_id + y_stride_id * MAT_B_X_LOADS_PER_THREAD;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;
            unsigned int act_data;
            if(std::is_same<inT,matT>::value && std::is_same<MAJOR,nvcuda::wmma::col_major>::value)
            {
                act_data = ((unsigned int *)data)[act_x + act_y * val_stride];
            }
            else
            {
                #pragma unroll
                for(unsigned int x_offset = 0; x_offset < MAT_B_X_VALUES_PER_REGISTER; x_offset++)
                {
                    unsigned int load_id = (std::is_same<MAJOR,nvcuda::wmma::col_major>::value) ? (x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER + act_y * stride) : ((x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER) * stride + act_y);
                    ((matT *)(&act_data))[x_offset] = (matT)(data[load_id]);
                }
            }

            #pragma unroll
            for(unsigned int copy_id = 0; copy_id < copy_per_thread; copy_id++)
            {
                ((unsigned int *)mat)[id + copy_id * copy_stride] = act_data;
            }
        }   
    }
}

template <typename mat_type, class DataManipulatorT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_b>::value, void>::type
load_matrix_sync(fragment<mat_type, M, N, K, matT>& mat, DataManipulatorT &data)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 4/sizeof(matT);
    //int val_stride = stride/MAT_B_X_VALUES_PER_REGISTER;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = K/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = N/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;
    constexpr unsigned int copy_stride = MAT_B_Y_LOADS_PER_THREAD*MAT_B_X_LOADS_PER_THREAD;
    constexpr unsigned int copy_per_thread = (std::is_same<half,matT>::value && (K == 16) && (M * N==256)) ? (8/copy_stride) : 1;
    constexpr bool is_load_linear = std::is_same<typename DataManipulatorT::t::t1,matT>::value && std::is_same<MAJOR,nvcuda::wmma::col_major>::value;

    // load from matrix a to fragments - indexing hell
    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = x_stride_id + y_stride_id * MAT_B_X_LOADS_PER_THREAD;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;
            unsigned int act_data[mat.getNumComponents()];
            if(is_load_linear)
            //if(false)
            {
                act_data[0] = data.load4BData2D(act_x * MAT_B_X_VALUES_PER_REGISTER, act_y);
            }
            else
            {
                data.template loadNx4BData2D<matT, std::is_same<MAJOR,nvcuda::wmma::col_major>::value>(act_x * MAT_B_X_VALUES_PER_REGISTER, act_y, (typename componentTypes<matT>::t1 *)act_data);
            }

            #pragma unroll
            for(unsigned int element_id = 0; element_id < mat.getNumComponents(); element_id++)
            {
                #pragma unroll
                for(unsigned int copy_id = 0; copy_id < copy_per_thread; copy_id++)
                {
                    ((unsigned int *)(mat.x[element_id]))[id + copy_id * copy_stride] = act_data[element_id];
                }
            }
        }   
    }
}

template <typename mat_type, typename inT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_b>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
load_matrix_sync(CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat, inT *data, unsigned int stride)
{
    load_matrix_sync<mat_type, inT, matT, M, N, K, MAJOR>(mat.x[0], data, stride);
}

/*template <typename mat_type, class DataManipulatorT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::matrix_b>::value, void>::type
load_matrix_sync(CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat, DataManipulatorT &data_getter)
{
    load_matrix_sync<mat_type, DataManipulatorT, matT, M, N, K, MAJOR>(mat.x, data_getter);
}*/

// load matrix accumulator
template <typename mat_type, typename inT, typename matT, int M, int N, int K, nvcuda::wmma::layout_t MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
load_matrix_sync(matT *mat, inT *data, unsigned int stride)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 2;
    int val_stride = stride/MAT_B_X_VALUES_PER_REGISTER;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = N/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = M/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;

    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = (y_stride_id & 1) + x_stride_id * 2 + (y_stride_id >> 1) * MAT_B_X_LOADS_PER_THREAD * 2;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;
            typename componentTypes<matT>::t2 act_data;
            // fast path -> no conversion of datatypes or layouts for threads is needed
            if(std::is_same<inT,matT>::value && (MAJOR == nvcuda::wmma::mem_row_major))
            {
                act_data = ((typename componentTypes<matT>::t2 *)data)[act_x + act_y * val_stride];
            }
            // slow path -> reindexing datatypes and/or conversion layouts for threads is needed
            else
            {
                #pragma unroll
                for(unsigned int x_offset = 0; x_offset < MAT_B_X_VALUES_PER_REGISTER; x_offset++)
                {
                    unsigned int load_id = (MAJOR == nvcuda::wmma::mem_row_major) ? (x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER + act_y * stride) : ((x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER) * stride + act_y);
                    ((matT *)(&act_data))[x_offset] = (matT)(data[load_id]);
                }
            }

            ((typename componentTypes<matT>::t2 *)mat)[id] = act_data;
        }   
    }
}

// load matrix accumulator
template <typename mat_type, class DataManipulatorT, typename matT, int M, int N, int K, nvcuda::wmma::layout_t MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value, void>::type
load_matrix_sync(fragment<mat_type, M, N, K, matT>& mat, DataManipulatorT &data)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 2;
    //int val_stride = stride/MAT_B_X_VALUES_PER_REGISTER;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = N/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = M/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;
    constexpr bool is_load_linear = std::is_same<typename DataManipulatorT::t::t1,matT>::value && (MAJOR == nvcuda::wmma::mem_row_major);

    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = (y_stride_id & 1) + x_stride_id * 2 + (y_stride_id >> 1) * MAT_B_X_LOADS_PER_THREAD * 2;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;
            typename componentTypes<matT>::t2 act_data[mat.getNumComponents()];
            if(is_load_linear)
            {
                act_data[0] = data.load2Data2D(act_x * MAT_B_X_VALUES_PER_REGISTER, act_y);
            }
            else
            {
                data.template loadNxMData2D<matT, MAJOR == nvcuda::wmma::mem_row_major, 2>(act_x * MAT_B_X_VALUES_PER_REGISTER, act_y, (typename componentTypes<matT>::t1 *)act_data);
            }

            #pragma unroll
            for(unsigned int element_id = 0; element_id < mat.getNumComponents(); element_id++)
            {
                ((typename componentTypes<matT>::t2 *)(mat.x[element_id]))[id] = act_data[element_id];
            }
        }   
    }
}

template <typename mat_type, typename inT, typename matT, int M, int N, int K, nvcuda::wmma::layout_t MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
load_matrix_sync(CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat, inT *data, unsigned int stride)
{
    load_matrix_sync<mat_type, inT, matT, M, N, K, MAJOR>(mat.x[0], data, stride);
}

/*template <typename mat_type, class DataManipulatorT, typename matT, int M, int N, int K, typename MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value, void>::type
load_matrix_sync(CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat, DataManipulatorT &data_getter)
{
    load_matrix_sync<mat_type, DataManipulatorT, matT, M, N, K, MAJOR>(mat.x, data_getter);
}*/

// store matrix accumulator
template <typename mat_type, typename matT, typename outT, int M, int N, int K, nvcuda::wmma::layout_t MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
store_matrix_sync(outT *data, matT *mat, unsigned int stride)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 2;
    int val_stride = stride/MAT_B_X_VALUES_PER_REGISTER;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = N/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = M/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;

    // load from accumulator matrix to fragments - indexing hell
    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = (y_stride_id & 1) + x_stride_id * 2 + (y_stride_id >> 1) * MAT_B_X_LOADS_PER_THREAD * 2;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;
            typename componentTypes<matT>::t2 act_data = ((typename componentTypes<matT>::t2 *)mat)[id];
            // fast path -> no conversion of datatypes or layouts for threads is needed
            if(std::is_same<matT,outT>::value && (MAJOR == nvcuda::wmma::mem_row_major))
            {
                ((typename componentTypes<matT>::t2 *)data)[act_x + act_y * val_stride] = act_data;
            }
            // slow path -> reindexing datatypes and/or conversion layouts for threads is needed
            else
            {
                #pragma unroll
                for(unsigned int x_offset = 0; x_offset < MAT_B_X_VALUES_PER_REGISTER; x_offset++)
                {
                    unsigned int store_id = (MAJOR == nvcuda::wmma::mem_row_major) ? (x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER + act_y * stride) : ((x_offset + act_x * MAT_B_X_VALUES_PER_REGISTER) * stride + act_y);
                    data[store_id] = (outT)(((matT *)(&act_data))[x_offset]);
                }
            }
        }   
    }
}

template <typename mat_type, typename matT, class DataManipulatorT, int M, int N, int K, nvcuda::wmma::layout_t MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value, void>::type
store_matrix_sync(DataManipulatorT &data, fragment<mat_type, M, N, K, matT>& mat)
{
    constexpr unsigned int THREADS_PER_X = 4;
    constexpr unsigned int THREADS_PER_Y = 8;
    constexpr unsigned int MAT_B_X_VALUES_PER_REGISTER = 2;
    
    constexpr unsigned int MAT_B_X_LOADS_PER_THREAD = N/(THREADS_PER_X*MAT_B_X_VALUES_PER_REGISTER);
    constexpr unsigned int MAT_B_Y_LOADS_PER_THREAD = M/THREADS_PER_Y;
    unsigned int thread_in_warp = threadIdx.x%WARP_SIZE;
    unsigned int y_thread_start = thread_in_warp/THREADS_PER_X;
    unsigned int x_thread_start = thread_in_warp%THREADS_PER_X;

    constexpr bool is_load_linear = std::is_same<typename DataManipulatorT::t::t1,matT>::value && (MAJOR == nvcuda::wmma::mem_row_major);
    // load from accumulator matrix to fragments - indexing hell
    #pragma unroll
    for(unsigned int y_stride_id = 0; y_stride_id < MAT_B_Y_LOADS_PER_THREAD; y_stride_id++)
    {
        unsigned int act_y = y_thread_start + y_stride_id * THREADS_PER_Y;
        #pragma unroll
        for(unsigned int x_stride_id = 0; x_stride_id < MAT_B_X_LOADS_PER_THREAD; x_stride_id++)
        {
            unsigned int id;
            if(MAT_B_Y_LOADS_PER_THREAD > 1) id = (y_stride_id & 1) + x_stride_id * 2 + (y_stride_id >> 1) * MAT_B_X_LOADS_PER_THREAD * 2;
            else id = x_stride_id;

            unsigned int act_x = x_thread_start + x_stride_id * THREADS_PER_X;

            typename componentTypes<matT>::t2 act_data[mat.getNumComponents()];
            #pragma unroll
            for (unsigned int element_id = 0; element_id < mat.getNumComponents(); element_id++)
            {
                act_data[element_id] = ((typename componentTypes<matT>::t2*)(mat.x[element_id]))[id];
            }
            // fast path -> no conversion of datatypes or layouts for threads is needed
            //if(is_load_linear)
            if(is_load_linear)
            {
                data.store2Data2D(act_data[0], act_x * MAT_B_X_VALUES_PER_REGISTER, act_y);
            }
            // slow path -> reindexing datatypes and/or conversion layouts for threads is needed
            else
            {
                data.template storeNxMData2D<matT, MAJOR == nvcuda::wmma::mem_row_major, MAT_B_X_VALUES_PER_REGISTER>(act_x * MAT_B_X_VALUES_PER_REGISTER, act_y, (typename componentTypes<matT>::t1 *)act_data);
            }
        }   
    }
}

template <typename mat_type, typename matT, typename outT, int M, int N, int K, nvcuda::wmma::layout_t MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value && componentTypes<matT>::getNumComponents() == 1, void>::type
store_matrix_sync(outT *data, CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat, unsigned int stride)
{
    store_matrix_sync<mat_type, matT, outT, M, N, K, MAJOR>(data, mat.x[0], stride);
}

/*template <typename mat_type, typename matT, class DataManipulatorT, int M, int N, int K, nvcuda::wmma::layout_t MAJOR>
__device__ inline typename std::enable_if<std::is_same<mat_type, nvcuda::wmma::accumulator>::value, void>::type
store_matrix_sync(DataManipulatorT &data_getter, CudaTensorLib::fragment<mat_type, M, N, K, matT> &mat)
{
    store_matrix_sync<mat_type, matT, DataManipulatorT, M, N, K, MAJOR>(data_getter, mat.x);
}*/

template <typename T, int num_elements>
__device__ void fill_fragment(T *ptr, T data)
{
    #pragma unroll
    for(int i = 0; i < num_elements; i++)
    {
        ptr[i] = data;
    }
}

template <typename matAccT, typename matT, int M, int N, int K> __device__ void mma_sync(matAccT *d, matT *a, matT *b, matAccT *c);

template <> __device__ void mma_sync<half, half, 16, 16, 16>(half *d, half *a, half *b, half *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    unsigned int *c_ptr = (unsigned int *)c;
    unsigned int *d_ptr = (unsigned int *)d;
    asm("wmma.mma.sync.aligned.row.col.m16n16k16.f16.f16 {%0,%1,%2,%3}, {%4,%5,%6,%7,%8,%9,%10,%11}, {%12,%13,%14,%15,%16,%17,%18,%19}, {%20,%21,%22,%23};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]));
}

template <> __device__ void mma_sync<float, half, 16, 16, 16>(float *d, half *a, half *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = c;
    float *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m16n16k16.f32.f32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11,%12,%13,%14,%15}, {%16,%17,%18,%19,%20,%21,%22,%23}, {%24,%25,%26,%27,%28,%29,%30,%31};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]),"=f"(d_ptr[4]),"=f"(d_ptr[5]),"=f"(d_ptr[6]),"=f"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]), "f"(c_ptr[4]), "f"(c_ptr[5]), "f"(c_ptr[6]), "f"(c_ptr[7]));
}

template <> __device__ void mma_sync<half, half, 32, 8, 16>(half *d, half *a, half *b, half *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    unsigned int *c_ptr = (unsigned int *)c;
    unsigned int *d_ptr = (unsigned int *)d;
    asm("wmma.mma.sync.aligned.row.col.m32n8k16.f16.f16 {%0,%1,%2,%3}, {%4,%5,%6,%7,%8,%9,%10,%11}, {%12,%13,%14,%15,%16,%17,%18,%19}, {%20,%21,%22,%23};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]));
}

template <> __device__ void mma_sync<float, half, 32, 8, 16>(float *d, half *a, half *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = c;
    float *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m32n8k16.f32.f32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11,%12,%13,%14,%15}, {%16,%17,%18,%19,%20,%21,%22,%23}, {%24,%25,%26,%27,%28,%29,%30,%31};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]),"=f"(d_ptr[4]),"=f"(d_ptr[5]),"=f"(d_ptr[6]),"=f"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]), "f"(c_ptr[4]), "f"(c_ptr[5]), "f"(c_ptr[6]), "f"(c_ptr[7]));
}

template <> __device__ void mma_sync<half, half, 8, 32, 16>(half *d, half *a, half *b, half *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    unsigned int *c_ptr = (unsigned int *)c;
    unsigned int *d_ptr = (unsigned int *)d;
    asm("wmma.mma.sync.aligned.row.col.m8n32k16.f16.f16 {%0,%1,%2,%3}, {%4,%5,%6,%7,%8,%9,%10,%11}, {%12,%13,%14,%15,%16,%17,%18,%19}, {%20,%21,%22,%23};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]));
}

template <> __device__ void mma_sync<float, half, 8, 32, 16>(float *d, half *a, half *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = c;
    float *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m8n32k16.f32.f32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11,%12,%13,%14,%15}, {%16,%17,%18,%19,%20,%21,%22,%23}, {%24,%25,%26,%27,%28,%29,%30,%31};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]),"=f"(d_ptr[4]),"=f"(d_ptr[5]),"=f"(d_ptr[6]),"=f"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]), "f"(c_ptr[4]), "f"(c_ptr[5]), "f"(c_ptr[6]), "f"(c_ptr[7]));
}

template <> __device__ void mma_sync<half, half, 16, 8, 8>(half *d, half *a, half *b, half *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    unsigned int *c_ptr = (unsigned int *)c;
    unsigned int *d_ptr = (unsigned int *)d;
    asm("mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16 {%0,%1}, {%2,%3}, {%4}, {%5,%6};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]),
         "r"(b_ptr[0]), "r"(c_ptr[0]),
         "r"(c_ptr[1]));
}

template <> __device__ void mma_sync<float, half, 16, 8, 8>(float *d, half *a, half *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = (float *)c;
    float *d_ptr = (float *)d;
    asm("mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]),
         "r"(b_ptr[0]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]));
}





#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 750
template <> __device__ void mma_sync<int, unsigned char, 16, 16, 16>(int *d, unsigned char *a, unsigned char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m16n16k16.s32.u8.u8.s32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9}, {%10,%11}, {%12,%13,%14,%15,%16,%17,%18,%19};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]),"=r"(d_ptr[4]),"=r"(d_ptr[5]),"=r"(d_ptr[6]),"=r"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]),
         "r"(b_ptr[0]), "r"(b_ptr[1]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]), "r"(c_ptr[4]), "r"(c_ptr[5]), "r"(c_ptr[6]), "r"(c_ptr[7]));
}

template <> __device__ void mma_sync<int, char, 16, 16, 16>(int *d, char *a, char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m16n16k16.s32.s8.s8.s32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9}, {%10,%11}, {%12,%13,%14,%15,%16,%17,%18,%19};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]),"=r"(d_ptr[4]),"=r"(d_ptr[5]),"=r"(d_ptr[6]),"=r"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]),
         "r"(b_ptr[0]), "r"(b_ptr[1]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]), "r"(c_ptr[4]), "r"(c_ptr[5]), "r"(c_ptr[6]), "r"(c_ptr[7]));
}

template <> __device__ void mma_sync<int, unsigned char, 32, 8, 16>(int *d, unsigned char *a, unsigned char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m32n8k16.s32.u8.u8.s32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11}, {%12}, {%13,%14,%15,%16,%17,%18,%19,%20};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]),"=r"(d_ptr[4]),"=r"(d_ptr[5]),"=r"(d_ptr[6]),"=r"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]),
         "r"(b_ptr[0]), 
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]), "r"(c_ptr[4]), "r"(c_ptr[5]), "r"(c_ptr[6]), "r"(c_ptr[7]));
}

template <> __device__ void mma_sync<int, char, 32, 8, 16>(int *d, char *a, char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m32n8k16.s32.s8.s8.s32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11}, {%12}, {%13,%14,%15,%16,%17,%18,%19,%20};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]),"=r"(d_ptr[4]),"=r"(d_ptr[5]),"=r"(d_ptr[6]),"=r"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]),
         "r"(b_ptr[0]), 
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]), "r"(c_ptr[4]), "r"(c_ptr[5]), "r"(c_ptr[6]), "r"(c_ptr[7]));
}

template <> __device__ void mma_sync<int, unsigned char, 8, 32, 16>(int *d, unsigned char *a, unsigned char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m8n32k16.s32.u8.u8.s32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8}, {%9,%10,%11,%12}, {%13,%14,%15,%16,%17,%18,%19,%20};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]),"=r"(d_ptr[4]),"=r"(d_ptr[5]),"=r"(d_ptr[6]),"=r"(d_ptr[7]) :
         "r"(a_ptr[0]), 
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]), "r"(c_ptr[4]), "r"(c_ptr[5]), "r"(c_ptr[6]), "r"(c_ptr[7]));
}

template <> __device__ void mma_sync<int, char, 8, 32, 16>(int *d, char *a, char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m8n32k16.s32.s8.s8.s32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8}, {%9,%10,%11,%12}, {%13,%14,%15,%16,%17,%18,%19,%20};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]),"=r"(d_ptr[4]),"=r"(d_ptr[5]),"=r"(d_ptr[6]),"=r"(d_ptr[7]) :
         "r"(a_ptr[0]), 
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]), "r"(c_ptr[4]), "r"(c_ptr[5]), "r"(c_ptr[6]), "r"(c_ptr[7]));
}

template <> __device__ void mma_sync<int, unsigned char, 8, 8, 16>(int *d, unsigned char *a, unsigned char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("mma.sync.aligned.m8n8k16.row.col.s32.u8.u8.s32 {%0,%1}, {%2}, {%3}, {%4,%5};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]) :
         "r"(a_ptr[0]),
         "r"(b_ptr[0]),
         "r"(c_ptr[0]), "r"(c_ptr[1]));
}

template <> __device__ void mma_sync<int, char, 8, 8, 16>(int *d, char *a, char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("mma.sync.aligned.m8n8k16.row.col.s32.s8.s8.s32 {%0,%1}, {%2}, {%3}, {%4,%5};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]) :
         "r"(a_ptr[0]),
         "r"(b_ptr[0]),
         "r"(c_ptr[0]), "r"(c_ptr[1]));
}
#endif

#if /*!defined(__CUDA_ARCH__) ||*/ __CUDA_ARCH__ >= 800
template <> __device__ void mma_sync<half, half, 16, 8, 16>(half *d, half *a, half *b, half *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    unsigned int *c_ptr = (unsigned int *)c;
    unsigned int *d_ptr = (unsigned int *)d;
    asm("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0,%1}, {%2,%3,%4,%5}, {%6,%7}, {%8,%9};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]),
         "r"(b_ptr[0]), "r"(b_ptr[1]),
         "r"(c_ptr[0]), "r"(c_ptr[1]));
}

template <> __device__ void mma_sync<float, __nv_bfloat16, 16, 8, 16>(float *d, __nv_bfloat16 *a, __nv_bfloat16 *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = (float *)c;
    float *d_ptr = (float *)d;
    asm("mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]),
         "r"(b_ptr[0]), "r"(b_ptr[1]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]));
}

template <> __device__ void mma_sync<float, half, 16, 8, 16>(float *d, half *a, half *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = (float *)c;
    float *d_ptr = (float *)d;
    asm("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]),
         "r"(b_ptr[0]), "r"(b_ptr[1]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]));
}

template <> __device__ void mma_sync<int, unsigned char, 16, 8, 16>(int *d, unsigned char *a, unsigned char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("mma.sync.aligned.m16n8k16.row.col.s32.u8.u8.s32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]),
         "r"(b_ptr[0]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]));
}

template <> __device__ void mma_sync<int, char, 16, 8, 16>(int *d, char *a, char *b, int *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    int *c_ptr = c;
    int *d_ptr = d;
    asm("mma.sync.aligned.m16n8k16.row.col.s32.s8.s8.s32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};" :
        "=r"(d_ptr[0]),"=r"(d_ptr[1]),"=r"(d_ptr[2]),"=r"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]),
         "r"(b_ptr[0]),
         "r"(c_ptr[0]), "r"(c_ptr[1]), "r"(c_ptr[2]), "r"(c_ptr[3]));
}

template <> __device__ void mma_sync<float, __nv_bfloat16, 16, 8, 8>(float *d, __nv_bfloat16 *a, __nv_bfloat16 *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = (float *)c;
    float *d_ptr = (float *)d;
    asm("mma.sync.aligned.m16n8k8.row.col.f32.bf16.bf16.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]),
         "r"(b_ptr[0]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]));
}

template <> __device__ void mma_sync<float, precision::tf32, 16, 8, 8>(float *d, precision::tf32 *a, precision::tf32 *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = (float *)c;
    float *d_ptr = (float *)d;
    asm("mma.sync.aligned.m16n8k8.row.col.f32.bf16.bf16.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]),
         "r"(b_ptr[0]), "r"(b_ptr[1]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]));
}

template <> __device__ void mma_sync<float, __nv_bfloat16, 16, 16, 16>(float *d, __nv_bfloat16 *a, __nv_bfloat16 *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = c;
    float *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m16n16k16.f32.bf16.bf16.f32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11,%12,%13,%14,%15}, {%16,%17,%18,%19,%20,%21,%22,%23}, {%24,%25,%26,%27,%28,%29,%30,%31};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]),"=f"(d_ptr[4]),"=f"(d_ptr[5]),"=f"(d_ptr[6]),"=f"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]), "f"(c_ptr[4]), "f"(c_ptr[5]), "f"(c_ptr[6]), "f"(c_ptr[7]));
}

template <> __device__ void mma_sync<float, __nv_bfloat16, 32, 8, 16>(float *d, __nv_bfloat16 *a, __nv_bfloat16 *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = c;
    float *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m32n8k16.f32.bf16.bf16.f32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11,%12,%13,%14,%15}, {%16,%17,%18,%19,%20,%21,%22,%23}, {%24,%25,%26,%27,%28,%29,%30,%31};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]),"=f"(d_ptr[4]),"=f"(d_ptr[5]),"=f"(d_ptr[6]),"=f"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]), "f"(c_ptr[4]), "f"(c_ptr[5]), "f"(c_ptr[6]), "f"(c_ptr[7]));
}

template <> __device__ void mma_sync<float, __nv_bfloat16, 8, 32, 16>(float *d, __nv_bfloat16 *a, __nv_bfloat16 *b, float *c)
{
    unsigned int *a_ptr = (unsigned int *)a;
    unsigned int *b_ptr = (unsigned int *)b;
    float *c_ptr = c;
    float *d_ptr = d;
    asm("wmma.mma.sync.aligned.row.col.m8n32k16.f32.bf16.bf16.f32 {%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9,%10,%11,%12,%13,%14,%15}, {%16,%17,%18,%19,%20,%21,%22,%23}, {%24,%25,%26,%27,%28,%29,%30,%31};" :
        "=f"(d_ptr[0]),"=f"(d_ptr[1]),"=f"(d_ptr[2]),"=f"(d_ptr[3]),"=f"(d_ptr[4]),"=f"(d_ptr[5]),"=f"(d_ptr[6]),"=f"(d_ptr[7]) :
         "r"(a_ptr[0]), "r"(a_ptr[1]), "r"(a_ptr[2]), "r"(a_ptr[3]), "r"(a_ptr[4]), "r"(a_ptr[5]), "r"(a_ptr[6]), "r"(a_ptr[7]),
         "r"(b_ptr[0]), "r"(b_ptr[1]), "r"(b_ptr[2]), "r"(b_ptr[3]), "r"(b_ptr[4]), "r"(b_ptr[5]), "r"(b_ptr[6]), "r"(b_ptr[7]),
         "f"(c_ptr[0]), "f"(c_ptr[1]), "f"(c_ptr[2]), "f"(c_ptr[3]), "f"(c_ptr[4]), "f"(c_ptr[5]), "f"(c_ptr[6]), "f"(c_ptr[7]));
}
#endif

template <typename matAccT, typename matT, int M, int N, int K>
__device__ typename std::enable_if<matT::t::getNumComponents() == matAccT::t::getNumComponents(), void>::type
mma_sync(fragment<nvcuda::wmma::accumulator, M, N, K, matAccT>& d,
         fragment<nvcuda::wmma::matrix_a, M, N, K, matT>& a,
         fragment<nvcuda::wmma::matrix_b, M, N, K, matT>& b,
         fragment<nvcuda::wmma::accumulator, M, N, K, matAccT>& c)
{
    #pragma unroll
    for (unsigned int component_id = 0; component_id < matT::t::getNumComponents(); component_id++)
    {
        mma_sync<matAccT, matT, M, N, K>(d.x[component_id], a.x[component_id], b.x[component_id], c.x[component_id]);
    }
}

}
#endif